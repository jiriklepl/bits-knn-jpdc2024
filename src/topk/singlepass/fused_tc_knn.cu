#include <cstdint>
#include <stdexcept>

#include "bits/cuda_array.hpp"
#include "bits/cuda_knn.hpp"
#include "bits/hip_stream.hpp"
#include "bits/knn.hpp"
#include "bits/topk/singlepass/fused_tc_knn.hpp"
#include "bits/topk/singlepass/fused_tc_policy.hpp"

#include "bits/topk/bitonic_sort_regs.cuh"
#include "bits/topk/singlepass/fused_tc_kernel.cuh"

namespace
{

/** Auxiliary function to transform runtime variable values to template constants.
 */
template <typename Policy, std::int32_t K>
void run(fused_tc_kernel_runner<Policy>& kernel)
{
    if (kernel.block_size == 128)
    {
        kernel.template operator()<K, 128>();
    }
    else if (kernel.block_size == 256)
    {
        kernel.template operator()<K, 256>();
    }
    else if (kernel.block_size == 512)
    {
        kernel.template operator()<K, 512>();
    }
    else
    {
        throw std::runtime_error{"Unsupported block size: " + std::to_string(kernel.block_size)};
    }
}

/** Auxiliary function to transform runtime variable values to template constants.
 */
template <typename Policy>
void run(fused_tc_kernel_runner<Policy>& kernel)
{
    if (kernel.k == 2)
    {
        run<Policy, 2>(kernel);
    }
    else if (kernel.k == 4)
    {
        run<Policy, 4>(kernel);
    }
    else if (kernel.k == 8)
    {
        run<Policy, 8>(kernel);
    }
    else if (kernel.k == 16)
    {
        run<Policy, 16>(kernel);
    }
    else if (kernel.k == 32)
    {
        run<Policy, 32>(kernel);
    }
    else if (kernel.k == 64)
    {
        run<Policy, 64>(kernel);
    }
    else if (kernel.k == 128)
    {
        run<Policy, 128>(kernel);
    }
    else
    {
        throw std::runtime_error{"Unsupported k value: " + std::to_string(kernel.k)};
    }
}

} // namespace

template <typename Policy>
void fused_tc_knn<Policy>::initialize(const knn_args& args)
{
    // skip allocation in cuda_knn::initialize()
    knn::initialize(args);

    out_dist_gpu_ = cuda_array<float, 2>{{query_count(), k()}};
    out_label_gpu_ = cuda_array<std::int32_t, 2>{{query_count(), k()}};
    points_gpu_ = cuda_array<float, 2>{{point_count(), dim()}};
    queries_gpu_ = cuda_array<float, 2>{{query_count(), dim()}};

    const auto points = points_gpu_.view();
    const auto queries = queries_gpu_.view();

    hip_stream::make_default()
        .copy_to_gpu_async(points, args_.points)
        .copy_to_gpu_async(queries, args_.queries)
        .sync();

    const auto aligned_dim = (points.size(1) + DIM_TILE_SIZE - 1) / DIM_TILE_SIZE * DIM_TILE_SIZE;
    const auto point_tiles = (points.size(0) * aligned_dim + DIM_TILE_SIZE * POINT_TILE_SIZE - 1) /
                             (DIM_TILE_SIZE * POINT_TILE_SIZE);
    in_points_gpu_ = cuda_array<input_t, 2>({point_tiles, DIM_TILE_SIZE * POINT_TILE_SIZE});
    in_point_norms_gpu_ = cuda_array<float, 1>({points.size(0)});

    assert(aligned_dim == (queries.size(1) + DIM_TILE_SIZE - 1) / DIM_TILE_SIZE * DIM_TILE_SIZE);
    const auto query_tiles = (queries.size(0) * aligned_dim + DIM_TILE_SIZE * QUERY_TILE_SIZE - 1) /
                             (DIM_TILE_SIZE * QUERY_TILE_SIZE);
    in_queries_gpu_ = cuda_array<input_t, 2>({query_tiles, DIM_TILE_SIZE * QUERY_TILE_SIZE});
    in_query_norms_gpu_ = cuda_array<float, 1>({queries.size(0)});
}

template <typename Policy>
void fused_tc_knn<Policy>::selection()
{
    auto points = points_gpu_.view();
    auto queries = queries_gpu_.view();

    fused_tc_kernel_runner<Policy> kernel;
    kernel.points = points;
    kernel.queries = queries;

    kernel.in_points = in_points_gpu_.view();
    kernel.in_point_norms = in_point_norms_gpu_.view();
    kernel.in_queries = in_queries_gpu_.view();
    kernel.in_query_norms = in_query_norms_gpu_.view();

    kernel.out_dist = out_dist_gpu();
    kernel.out_label = out_label_gpu();

    kernel.k = k();
    kernel.block_size = selection_block_size();

    run(kernel);

    hip_stream::make_default().sync();
}

template void fused_tc_knn<fused_tc_kernel_half_policy>::initialize(const knn_args& args);
template void fused_tc_knn<fused_tc_kernel_half_policy>::selection();

template void fused_tc_knn<fused_tc_kernel_bfloat16_policy>::initialize(const knn_args& args);
template void fused_tc_knn<fused_tc_kernel_bfloat16_policy>::selection();

template void fused_tc_knn<fused_tc_kernel_double_policy>::initialize(const knn_args& args);
template void fused_tc_knn<fused_tc_kernel_double_policy>::selection();
