#include <cstddef>
#include <cstdint>
#include <stdexcept>
#include <string>
#include <vector>

#include "bits/hip_stream.hpp"
#include "bits/topk/singlepass/fused_cache_kernel.hpp"
#include "bits/topk/singlepass/fused_cache_knn.hpp"

#include "bits/topk/singlepass/fused_cache_kernel_structs.cuh"

namespace
{

struct fused_cache
{
    const float* queries;
    const float* db;
    std::int32_t dim;
    std::int32_t num_queries;
    std::int32_t num_db;
    float* out_dist;
    std::int32_t* out_label;

    template <std::int32_t QUERY_REG, std::int32_t DB_REG, std::int32_t DIM_REG,
              std::int32_t BLOCK_QUERY_DIM, std::int32_t DIM_MULT, std::int32_t K>
    void run()
    {
        constexpr std::int32_t BLOCK_SIZE = 256;
        constexpr std::int32_t BLOCK_DB_DIM = BLOCK_SIZE / BLOCK_QUERY_DIM;
        constexpr std::int32_t DIM_TILE = DIM_MULT * DIM_REG;
        constexpr std::int32_t QUERIES_PER_BLOCK = BLOCK_QUERY_DIM * QUERY_REG;

        using kernel_t = fused_cache_kernel<QUERY_REG, DB_REG, DIM_REG, BLOCK_QUERY_DIM,
                                            BLOCK_DB_DIM, DIM_MULT, K>;

        kernel_t kernel{.tmp_storage = nullptr,
                        .queries = queries,
                        .db = db,
                        .dim = dim,
                        .num_queries = num_queries,
                        .num_db = num_db,
                        .out_dist = out_dist,
                        .out_label = out_label};

        const dim3 block(BLOCK_QUERY_DIM, BLOCK_DB_DIM, 1);
        const dim3 grid((num_queries + QUERIES_PER_BLOCK - 1) / QUERIES_PER_BLOCK, 1, 1);

        if (dim < DIM_TILE)
        {
            throw std::runtime_error{"Dimension of vectors must not be lower than " +
                                     std::to_string(DIM_TILE) + ", but is " + std::to_string(dim)};
        }

        launch_fused_cache(kernel, grid, block);
    }

    template <std::int32_t QUERY_REG, std::int32_t DB_REG, std::int32_t DIM_REG,
              std::int32_t QUERY_BLOCK_DIM, std::int32_t DIM_MULT>
    void run(std::int32_t k)
    {
        if (k == 4)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 4>();
        }
        else if (k == 8)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 8>();
        }
        else if (k == 16)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 16>();
        }
        else if (k == 32)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 32>();
        }
        else if (k == 64)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 64>();
        }
        else if (k == 128)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, DIM_MULT, 128>();
        }
        else
        {
            throw std::runtime_error{"Unsupported k value: " + std::to_string(k)};
        }
    }

    template <std::int32_t QUERY_REG, std::int32_t DB_REG, std::int32_t DIM_REG,
              std::int32_t QUERY_BLOCK_DIM>
    void run(std::int32_t dim_mult, std::int32_t k)
    {
        if (dim_mult == 1)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, 1>(k);
        }
        else if (dim_mult == 2)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, 2>(k);
        }
        else if (dim_mult == 4)
        {
            run<QUERY_REG, DB_REG, DIM_REG, QUERY_BLOCK_DIM, 4>(k);
        }
        else
        {
            throw std::runtime_error{"Unsupported dim_mult value: " + std::to_string(dim_mult)};
        }
    }

    template <std::int32_t QUERY_REG, std::int32_t DB_REG, std::int32_t DIM_REG>
    void run(std::int32_t query_block_size, std::int32_t dim_mult, std::int32_t k)
    {
        if (query_block_size == 1)
        {
            run<QUERY_REG, DB_REG, DIM_REG, 1>(dim_mult, k);
        }
        else if (query_block_size == 2)
        {
            run<QUERY_REG, DB_REG, DIM_REG, 2>(dim_mult, k);
        }
        else if (query_block_size == 4)
        {
            run<QUERY_REG, DB_REG, DIM_REG, 4>(dim_mult, k);
        }
        else
        {
            throw std::runtime_error{"Unsupported query_block_size value: " +
                                     std::to_string(query_block_size)};
        }
    }

    template <std::int32_t QUERY_REG, std::int32_t DB_REG>
    void run(std::int32_t dim_reg, std::int32_t query_block_size, std::int32_t dim_mult,
             std::int32_t k)
    {
        if (dim_reg == 1)
        {
            run<QUERY_REG, DB_REG, 1>(query_block_size, dim_mult, k);
        }
        else if (dim_reg == 2)
        {
            run<QUERY_REG, DB_REG, 2>(query_block_size, dim_mult, k);
        }
        else if (dim_reg == 4)
        {
            run<QUERY_REG, DB_REG, 4>(query_block_size, dim_mult, k);
        }
        else
        {
            throw std::runtime_error{"Unsupported dim_reg value: " + std::to_string(dim_reg)};
        }
    }

    template <std::int32_t QUERY_REG>
    void run(std::int32_t db_reg, std::int32_t dim_reg, std::int32_t query_block_size,
             std::int32_t dim_mult, std::int32_t k)
    {
        if (db_reg == 4)
        {
            run<QUERY_REG, 4>(dim_reg, query_block_size, dim_mult, k);
        }
        else if (db_reg == 8)
        {
            run<QUERY_REG, 8>(dim_reg, query_block_size, dim_mult, k);
        }
        else if (db_reg == 16)
        {
            run<QUERY_REG, 16>(dim_reg, query_block_size, dim_mult, k);
        }
        else
        {
            throw std::runtime_error{"Unsupported db_reg value: " + std::to_string(db_reg)};
        }
    }

    void run(std::int32_t queries_reg, std::int32_t db_reg, std::int32_t dim_reg,
             std::int32_t query_block_size, std::int32_t dim_mult, std::int32_t k)
    {
        if (queries_reg == 2)
        {
            run<2>(db_reg, dim_reg, query_block_size, dim_mult, k);
        }
        else if (queries_reg == 4)
        {
            run<4>(db_reg, dim_reg, query_block_size, dim_mult, k);
        }
        else if (queries_reg == 8)
        {
            run<8>(db_reg, dim_reg, query_block_size, dim_mult, k);
        }
        else if (queries_reg == 16)
        {
            run<16>(db_reg, dim_reg, query_block_size, dim_mult, k);
        }
        else
        {
            throw std::runtime_error{"Unsupported queries_reg value: " + std::to_string(queries_reg)};
        }
    }
};

} // namespace

void fused_cache_knn::initialize(const knn_args& args)
{
    // skip allocation in cuda_knn::initialize()
    knn::initialize(args);

    out_dist_gpu_ = cuda_array<float, 2>{{query_count(), k()}};
    out_label_gpu_ = cuda_array<std::int32_t, 2>{{query_count(), k()}};
    points_gpu_ = cuda_array<float, 2>{{dim(), point_count()}};
    queries_gpu_ = cuda_array<float, 2>{{query_count(), dim()}};

    // transpose the DB matrix if necessary
    auto points = args_.points;
    std::vector<float> points_transposed;
    if (args.points_layout != matrix_layout::column_major)
    {
        points_transposed.resize(points_gpu_.view().size());
        for (std::size_t i = 0; i < point_count(); ++i)
        {
            for (std::size_t j = 0; j < dim(); ++j)
            {
                points_transposed[j * point_count() + i] = args_.points[i * dim() + j];
            }
        }
        points = points_transposed.data();
    }

    // transpose the query matrix if necessary
    auto queries = args_.queries;
    std::vector<float> queries_transposed;
    if (args.queries_layout != matrix_layout::column_major)
    {
        queries_transposed.resize(queries_gpu_.view().size());
        for (std::size_t i = 0; i < query_count(); ++i)
        {
            for (std::size_t j = 0; j < dim(); ++j)
            {
                queries_transposed[j * query_count() + i] = args_.queries[i * dim() + j];
            }
        }
        queries = queries_transposed.data();
    }

    hip_stream::make_default()
        .copy_to_gpu_async(points_gpu_.view(), points)
        .copy_to_gpu_async(queries_gpu_.view(), queries)
        .sync();
}

void fused_cache_knn::distances()
{
    // no computation
}

void fused_cache_knn::selection()
{
    fused_cache kernel{.queries = queries_gpu_.view().data(),
                       .db = points_gpu_.view().data(),
                       .dim = (std::int32_t)args_.dim,
                       .num_queries = (std::int32_t)query_count(),
                       .num_db = (std::int32_t)point_count(),
                       .out_dist = out_dist_gpu_.view().data(),
                       .out_label = out_label_gpu_.view().data()};

    kernel.run(args_.items_per_thread[0], args_.items_per_thread[1], args_.items_per_thread[2],
               args_.selection_block_size, args_.deg, k());

    hip_stream::make_default().sync();
}
